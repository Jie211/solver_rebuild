#include "cuda_func.cuh"

void malloc_cuda_1d(int num_x, double *arr)
{
  hipMalloc((void**)&arr, num_x*sizeof(double));
}

void malloc_cuda_1i(int num_x, int *arr)
{
  hipMalloc((void**)&arr, num_x*sizeof(int));
}

void free_cuda_1d(double *arr)
{
  hipFree(arr);
}

void free_cuda_1i(int *arr)
{
  hipFree(arr);
}

